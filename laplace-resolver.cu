#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#define THREADMAX 1000



#define ROWS 10
#define COLUMNS 10
#define PRECISION 1e-9
#define RANGE 100


__host__ int checkConvergence(int* arr, int size){

    int sum = 0;

    for(int i=0; i<size; i++)
        sum += arr[i];

    return sum;
            
}

__global__ void jacobi(double* std_vector, double* out_vector, int* er, int size, int blockSize){

    
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    

    if(globalIndex < size){

    int thIdx = threadIdx.x;
    int row = globalIndex/ROWS;
    int col = (fmod((double)globalIndex/COLUMNS, 1) * COLUMNS) + 1e-9;
    extern __shared__ int redc[];
    double jacob_val = 0;
    
    
    redc[thIdx] = 0;

    
    if(row > 0 && col < COLUMNS-1 && row < ROWS-1 && col > 0){

        jacob_val = 0.25 * (
            std_vector[(row-1)*COLUMNS+col] +
            std_vector[row*COLUMNS+col+1] +
            std_vector[(row+1)*COLUMNS+col] +
            std_vector[row*COLUMNS+col-1]
        );

        out_vector[row*COLUMNS+col] = jacob_val;


        if (fabsf(jacob_val - std_vector[row*COLUMNS+col]) < PRECISION)
            redc[thIdx] = 1;
 
    }

    __syncthreads();

    for (int s = blockSize/2; s>0; s/=2) {
        if (thIdx<s)
            redc[thIdx] += redc[thIdx+s];
        __syncthreads();
    }

    printf(" \n redc[%d] = %d \n", thIdx, redc[thIdx]);


    if(threadIdx.x == 0)
        er[blockIdx.x] = redc[0];       

    }


}

void init(double* m);
void initEr(int* er, int blockAmount);


int main(int aArgc, char* aArgv[])
{

    double *matrix, *j_matrix;
    double *d_matrix, *d_j_matrix;
    int *er, *d_er;

    long int size = ROWS * COLUMNS * sizeof(double);
    int erSize = 0;

    srand(time(NULL));


    matrix = (double*)malloc(size);

    j_matrix = (double*)malloc(size);


    int blockSize = COLUMNS < THREADMAX ? COLUMNS : THREADMAX; 
    int blockAmount = ROWS*(COLUMNS/THREADMAX <= 0 ? 1 : COLUMNS/THREADMAX) + 
                    ceil((ceil(COLUMNS/THREADMAX) - COLUMNS/THREADMAX));

    erSize = sizeof(int) * blockAmount;
    er = (int*)malloc(erSize);

    init(matrix);
    initEr(er, blockAmount);

    printf("\n %d %d \n", blockSize, blockAmount);
    

 
    hipMalloc((void**) &d_matrix, size);
    hipMalloc((void**) &d_j_matrix, size);
    hipMalloc((void**) &d_er, erSize);
    

    hipMemcpy(d_er, er, erSize, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);
    hipMemcpy(d_j_matrix, matrix, size, hipMemcpyHostToDevice);

    int counter = 0;
    

    while(checkConvergence(er, blockAmount) <= 0){


        jacobi<<<blockAmount, THREADMAX, blockSize>>>
        (d_matrix, d_j_matrix, d_er, ROWS*COLUMNS, blockSize);
        hipDeviceSynchronize();

        hipMemcpy(d_matrix, d_j_matrix, size, hipMemcpyDeviceToDevice);
        hipMemcpy(er, d_er, erSize, hipMemcpyDeviceToHost);
       
        counter++;
    }

    printf("\n Counter = %d \n", counter);


    hipMemcpy(j_matrix, d_j_matrix, size, hipMemcpyDeviceToHost);


    printf("\n\n");

    // for(int i=0; i<ROWS; i++){
    //     for(int j=0; j<COLUMNS; j++){
    //         printf("%.2f ", j_matrix[i*COLUMNS+j]);
    //     }
    //     printf("\n");
    // }


    free(matrix);
    free(j_matrix);
    free(er);

    hipFree(d_matrix);
    hipFree(d_j_matrix);
    hipFree(d_er);


    return 0;
}

void init(double* m){

    //read only values
    double r = 0;
    for(int i=0; i<ROWS; i++){
        for(int j=0; j<COLUMNS; j++){

            r = (double)(rand()%RANGE);
            if(i == 0 || j == 0 || j == COLUMNS-1 || i == ROWS-1)
                m[i*COLUMNS+j] = r;
            else
                m[i*COLUMNS+j] = r;

            // e[i*COLUMNS+j] = 0;
            // printf("%f ", m[i*COLUMNS+j]);
        }
        // printf("\n");
    }

}

void initEr(int* er, int blockAmount){

    for(int i=0;i<blockAmount; i++)
        er[i] = 0;
}